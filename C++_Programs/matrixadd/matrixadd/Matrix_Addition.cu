#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "../common/book.h"
#include <hip/hip_runtime.h>

#define M 3
#define N 3

/*Defining kernel function matrixAdd which takes 3 arguments A,B,C these are matrix of 3*3 A & B are input matrix and matrix C is the sum of A & B */

__global__ void matrixAdd(float *A, float *B, float *C) {
	int index = threadIdx.x+ blockDim.x * blockIdx.x; 

	/*The sum of matrix A & B is stored in C*/

	C[index] = A[index] + B[index]; 

	//printf("ThreadIdx.x : %d\tblockDim.x : %d\tblockIdx.x : %d ThreadIdx.y : %d\tblockDim.y : %d\tblockIdx.y : %d index: %d\n", threadIdx.x, blockDim.x, blockIdx.x, threadIdx.y, blockDim.y, blockIdx.y, index);

}


int main() {

	float A[M][N], B[M][N],C[M][N]; /*Declaring matrix A & B of size M*N */
	int i, j;
	float *dev_a,*dev_b,*dev_c;

	/*--------Taking user input for matrix A elements----------*/

	printf("\n Enter elements of first matrix A of size %d * %d\n",M,N);
	for (i = 0; i<M; i++) // i is representing row of matrix 
	{
		for (j = 0; j<N; j++) // j is representing column of matrix 
		{
			printf("Enter the element A[%d][%d] : ", i, j);
			scanf("%f", &A[i][j]); 
		}
	}


	/*--------Taking user input for matrix B elements----------*/

	printf("\n Enter elements of second matrix B of size %d * %d\n",M,N);
	for(i = 0; i<M; i++)
	{
		for (j = 0; j<N; j++)
		{
			printf("Enter the element B[%d][%d] : ", i, j);
			scanf("%f", &B[i][j]);
		}
	}
	/*--------Allocating memory in GPU by using hipMalloc----------*/

	hipMalloc((void**)&dev_a, (M*N) * sizeof(float));
	hipMalloc((void**)&dev_b, (M*N) * sizeof(float));
	hipMalloc((void**)&dev_c, (M*N) * sizeof(float));

	/*--------Copying elements of 2D matrix A, B from host(CPU) to device(GPU) by using hipMemcpy----------*/

	hipMemcpy(dev_a, A, (M*N) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, B, (M*N) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, C, (M*N) * sizeof(float), hipMemcpyHostToDevice);

	/*---------Calling kernel function-------------*/

	matrixAdd<<< M,N >>>(dev_a, dev_b, dev_c); //  Calling kernel function with 3 block and 3 threads per block

	hipDeviceSynchronize(); // Synchronizing CPU with GPU

	/*-------- After the GPU kernel function executes it copies the result stored in device to host i.e CPU */

	hipMemcpy(C, dev_c, (M*N) * sizeof(float), hipMemcpyDeviceToHost);
	
	/*-----------Printing the addition of two matrix A & B---------------*/
	printf("\n");
	printf("Sum of two matices :\n ");

	for(i = 0; i<M; i++) 
	{
		for (j = 0; j<N; j++) 
		{
			printf("%.2f\t ",C[i][j]);
		}
		printf("\n");
	}

	// Free the memory allocated in GPU

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}